#include "hip/hip_runtime.h"
// corresponded header file
// necessary project headers
#include "DeconvLRCore.cuh"
#include "Helper.cuh"
// 3rd party libraries headers
#include <hip/hip_runtime.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hipfft/hipfft.h>
// standard libraries headers
#include <cstdint>
// system headers

#define DIVUP(x, y) ((x+y-1)/y)

namespace PSF {

// deviated PSF
hipArray_t d_psfDev = nullptr;
texture<float, hipTextureType3D, hipReadModeElementType> psfTexRef;

namespace {
__global__
void createGrid_kernel(
    int3 *d_grid,
    const size_t nx, const size_t ny, const size_t nz
) {
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;

    // skip out-of-bound threads
    if (ix >= nx or iy >= ny or iz >= nz) {
        return;
    }

    int idx = iz * (nx*ny) + iy * nx + ix;
    d_grid[idx] = make_int3(ix, iy, iz);
}

struct MultiplyWeighting
    : public thrust::unary_function<int3, float4> {
    MultiplyWeighting(
        const float *data,
        const size_t nx_, const size_t ny_, const size_t nz_
    )
        : d_weight(data), nx(nx_), ny(ny_), nz(nz_) {
    }

    __host__ __device__
    float4 operator()(const int3 &p) const {
        const int idx = p.z * (nx*ny) + p.y * nx + p.x;
        const float w = d_weight[idx];
        return make_float4(p.x*w, p.y*w, p.z*w, w);
    }

private:
    const float *d_weight;
    size_t nx, ny, nz;
};

__global__
void alignCenter_kernel(
    float *d_odata,
    const size_t nx, const size_t ny, const size_t nz,
    const float ox, const float oy, const float oz
) {
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;

    // skip out-of-bound threads
    if (ix >= nx or iy >= ny or iz >= nz) {
        return;
    }

    // sampling from the texture
    // (coordinates are backtracked to the deviated ones)
    int idx = iz * (nx*ny) + iy * nx + ix;
    d_odata[idx] = tex3D(psfTexRef, ix+ox+0.5f, iy+oy+0.5f, iz+oz+0.5f);
}
}

float3 findCentroid(
    float *h_psf,
    const size_t nx, const size_t ny, const size_t nz
) {
    // pinned down the host memory region
    float *d_psf;
    const size_t nelem = nx * ny * nz;
    cudaErrChk(hipHostRegister(
        h_psf,
        nelem * sizeof(float),
        hipHostRegisterMapped
    ));
    cudaErrChk(hipHostGetDevicePointer(&d_psf, h_psf, 0));

    // create a 3-D grid for weighting
    int3 *d_grid;
    cudaErrChk(hipMalloc(&d_grid, nelem * sizeof(int3)));
    dim3 nthreads(16, 16, 4);
    dim3 nblocks(
        DIVUP(nx, nthreads.x), DIVUP(ny, nthreads.y), DIVUP(nz, nthreads.z)
    );
    createGrid_kernel<<<nblocks, nthreads>>>(d_grid, nx, ny, nz);
    cudaErrChk(hipPeekAtLastError());

    // calculate the centroid along grid points
    float4 result = thrust::transform_reduce(
        thrust::device,
        d_grid, d_grid + nelem,
        MultiplyWeighting(d_psf, nx, ny, nz),
        make_float4(0, 0, 0, 0),
        thrust::plus<float4>()
    );

    float3 centroid = make_float3(
        result.x/result.w, result.y/result.w, result.z/result.w
    );

    // release the resources
    cudaErrChk(hipFree(d_grid));
    cudaErrChk(hipHostUnregister(h_psf));

    return centroid;
}

void bindData(
    float *h_psf,
    const size_t nx, const size_t ny, const size_t nz
) {
    // create hipArray for the texture
    hipChannelFormatDesc desc = hipCreateChannelDesc(
        32, 0, 0, 0, hipChannelFormatKindFloat
    );
    hipExtent extent = make_hipExtent(
        nx, ny, nz
    );
    cudaErrChk(hipMalloc3DArray(
        &d_psfDev,
        &desc,
        extent,
        hipArrayDefault
    ));

    // copy data from host to device
    hipMemcpy3DParms parms = {0};
    parms.srcPtr = make_hipPitchedPtr(
        h_psf,
        nx * sizeof(float), nx, ny
    );
    parms.dstArray = d_psfDev;
    parms.extent = extent;
    parms.kind = hipMemcpyHostToDevice;
    cudaErrChk(hipMemcpy3D(&parms));

    // texture coordinates are not normalized
    psfTexRef.normalized = false;
    // sampled data is interpolated
    psfTexRef.filterMode = hipFilterModeLinear;
    // wrap around the texture if exceeds border limit
    psfTexRef.addressMode[0] = hipAddressModeWrap;
    psfTexRef.addressMode[1] = hipAddressModeWrap;
    psfTexRef.addressMode[2] = hipAddressModeWrap;

    // bind the texture
    cudaErrChk(hipBindTextureToArray(
        psfTexRef,  // texture to bind
        d_psfDev,   // memory array on device
        desc        // channel format
    ));
}

void alignCenter(
    float *h_psf,
    const size_t nx, const size_t ny, const size_t nz,
    const float3 centroid
) {
    // coordinate of the center of the volume
    const float3 center = make_float3(
        (nx-1)/2.0f, (ny-1)/2.0f, (nz-1)/2.0f
    );
    // offset
    const float3 offset = centroid - center;

    fprintf(stderr, "[DEBUG] offset = (%.2f, %.2f, %.2f)\n", offset.x, offset.y, offset.z);

    // pinned down the host memory region
    float *d_psf;
    cudaErrChk(hipHostRegister(
        h_psf,
        nx * ny * nz * sizeof(float),
        hipHostRegisterMapped
    ));
    cudaErrChk(hipHostGetDevicePointer(&d_psf, h_psf, 0));

    // begin resample the kernel
    dim3 nthreads(16, 16, 4);
    dim3 nblocks(
        DIVUP(nx, nthreads.x), DIVUP(ny, nthreads.y), DIVUP(nz, nthreads.z)
    );
    alignCenter_kernel<<<nblocks, nthreads>>>(
        d_psf,
        nx, ny, nz,
        offset.x, offset.y, offset.z
    );
    cudaErrChk(hipPeekAtLastError());

    // release the resources
    cudaErrChk(hipHostUnregister(h_psf));
}

void release() {
    // unbind the texture
    cudaErrChk(hipUnbindTexture(psfTexRef));
    cudaErrChk(hipFreeArray(d_psfDev));
}

}

namespace OTF {

hipArray_t d_otf = nullptr;
texture<hipfftComplex, hipTextureType3D, hipReadModeElementType> otfTexRef;

namespace {

}

void calculate(
    float *h_psf,
    const size_t nx, const size_t ny, const size_t nz
) {
    // pinned down the host memory region
    float *d_psf;
    const size_t nelem = nx * ny * nz;
    cudaErrChk(hipHostRegister(
        h_psf,
        nelem * sizeof(float),
        hipHostRegisterMapped
    ));
    cudaErrChk(hipHostGetDevicePointer(&d_psf, h_psf, 0));

    // create FFT plan
    hipfftHandle otfHdl;
    cudaErrChk(hipfftPlan3d(
        &otfHdl,
        nz, ny, nx,
        HIPFFT_R2C
    ));
    // estimate resource requirements
    size_t wsSz;
    cudaErrChk(hipfftGetSize3d(
        otfHdl,
        nz, ny, nx,
        HIPFFT_R2C,
        &wsSz
    ));
    fprintf(stderr, "[DEBUG] PSF -> OTF requires %ld bytes\n", wsSz);

    // allocate device memory to buffer the result
    hipfftComplex *d_otfTpl;
    cudaErrChk(hipMalloc(
        &d_otfTpl,
        nx * ny * (nz/2+1) * sizeof(hipfftComplex)
    ));

    // begin PSF to OTF
    cudaErrChk(hipfftExecR2C(otfHdl, d_psf, d_otfTpl));

    // release the resources
    cudaErrChk(hipfftDestroy(otfHdl));
    cudaErrChk(hipHostUnregister(h_psf));
}

void interpolate() {
}

void release() {
}

}

namespace Kernel {

texture<hipfftComplex, 2, hipReadModeElementType> otfTex;

template <typename T_out, typename T_in>
__global__
void convertTypeKernel(T_out *dst, T_in *src,
                       const int nx, const int ny, const int nz,
                       const size_t pitchDst, const size_t pitchSrc) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < nx * ny * nz;
         i += blockDim.x * gridDim.x) {
        int z = i / nx / ny;
        int y = (i / nx) % ny;
        int x = i % nx;
        int nPitchSrc = pitchSrc / sizeof(T_in);
        int nPitchDst = pitchDst / sizeof(T_out);
        T_in* ptrSrc = src + (z * ny + y) * nPitchSrc + x;
        T_out* ptrDst = dst + (z * ny + y) * nPitchDst + x;
        *ptrDst = (T_out)(*ptrSrc);
    }
}

template <typename T_out, typename T_in>
__host__
void convertType(T_out *dst, T_in *src,
                 const hipExtent extDst, const hipExtent extSrc) {
    int nSMs;
    hipDeviceGetAttribute(&nSMs, hipDeviceAttributeMultiprocessorCount, 0);
    int nx = extSrc.width / sizeof(T_in);
    int ny = extSrc.height;
    int nz = extSrc.depth;
    convertTypeKernel<T_out, T_in><<<32*nSMs, 256>>>(dst, src, nx, ny, nz,
                                                     extDst.width, extSrc.width);
}

/*
 * Explicit instantiation
 */
template void convertType(hipfftReal *dst, uint16_t *src,
                          const hipExtent extDst, const hipExtent extSrc);

}
