#include "hip/hip_runtime.h"
// corresponded header file
// necessary project headers
#include "DeconvLRCore.cuh"
#include "Helper.cuh"
// 3rd party libraries headers
#include <hip/hip_runtime.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hipfft/hipfft.h>
// standard libraries headers
#include <cstdint>
// system headers

#define DIVUP(x, y) ((x+y-1)/y)

namespace PSF {

// deviated PSF
hipArray_t d_psfDev = nullptr;
texture<float, hipTextureType3D, hipReadModeElementType> psfTexRef;

namespace {
__global__
void createGrid_kernel(
    int3 *d_grid,
    const size_t nx, const size_t ny, const size_t nz
) {
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;

    // skip out-of-bound threads
    if (ix >= nx or iy >= ny or iz >= nz) {
        return;
    }

    int idx = iz * (nx*ny) + iy * nx + ix;
    d_grid[idx] = make_int3(ix, iy, iz);
}

struct MultiplyWeighting
    : public thrust::unary_function<int3, float4> {
    MultiplyWeighting(
        const float *data,
        const size_t nx_, const size_t ny_, const size_t nz_
    )
        : d_weight(data), nx(nx_), ny(ny_), nz(nz_) {
    }

    __host__ __device__
    float4 operator()(const int3 &p) const {
        const int idx = p.z * (nx*ny) + p.y * nx + p.x;
        const float w = d_weight[idx];
        return make_float4(p.x*w, p.y*w, p.z*w, w);
    }

private:
    const float *d_weight;
    size_t nx, ny, nz;
};

struct WeightedSum
    : public thrust::binary_function<float4, float4, float4> {
    __host__ __device__
    float4 operator()(const float4 &a, const float4 &b) const {
        return make_float4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
    }
};
}

float3 findCentroid(
    float *h_psf,
    const size_t nx, const size_t ny, const size_t nz
) {
    // pinned down the host memory region
    float *d_psf;
    const size_t nelem = nx * ny * nz;
    cudaErrChk(hipHostRegister(
        h_psf,
        nelem * sizeof(float),
        hipHostRegisterMapped
    ));
    cudaErrChk(hipHostGetDevicePointer(&d_psf, h_psf, 0));

    // create a 3-D grid for weighting
    int3 *d_grid;
    cudaErrChk(hipMalloc(&d_grid, nelem * sizeof(int3)));
    dim3 nthreads(16, 16, 4);
    dim3 nblocks(
        DIVUP(nx, nthreads.x), DIVUP(ny, nthreads.y), DIVUP(nz, nthreads.z)
    );
    createGrid_kernel<<<nblocks, nthreads>>>(d_grid, nx, ny, nz);
    cudaErrChk(hipPeekAtLastError());

    // calculate the centroid along grid points
    float4 result = thrust::transform_reduce(
        thrust::device,
        d_grid, d_grid + nelem,
        MultiplyWeighting(d_psf, nx, ny, nz),
        make_float4(0, 0, 0, 0),
        WeightedSum()
    );

    float3 centroid = make_float3(
        result.x/result.w, result.y/result.w, result.z/result.w
    );

    // release the resources
    cudaErrChk(hipFree(d_grid));
    cudaErrChk(hipHostUnregister(h_psf));

    return centroid;
}

void bindData(
    float *h_psf,
    const size_t nx, const size_t ny, const size_t nz
) {
    // create hipArray for the texture
    hipChannelFormatDesc desc = hipCreateChannelDesc(
        32, 0, 0, 0, hipChannelFormatKindFloat
    );
    hipExtent extent = make_hipExtent(
        nx, ny, nz
    );
    cudaErrChk(hipMalloc3DArray(
        &d_psfDev,
        &desc,
        extent,
        hipArrayDefault
    ));

    // copy data from host to device
    hipMemcpy3DParms parms = {0};
    parms.srcPtr = make_hipPitchedPtr(
        h_psf,
        nx * sizeof(float), nx, ny
    );
    parms.dstArray = d_psfDev;
    parms.extent = extent;
    parms.kind = hipMemcpyHostToDevice;
    cudaErrChk(hipMemcpy3D(&parms));

    // texture coordinates are not normalized
    psfTexRef.normalized = false;
    // sampled data is interpolated
    psfTexRef.filterMode = hipFilterModeLinear;
    // wrap around the texture if exceeds border limit
    psfTexRef.addressMode[0] = hipAddressModeWrap;
    psfTexRef.addressMode[1] = hipAddressModeWrap;
    psfTexRef.addressMode[2] = hipAddressModeWrap;

    // bind the texture
    cudaErrChk(hipBindTextureToArray(
        psfTexRef,  // texture to bind
        d_psfDev,   // memory array on device
        desc        // channel format
    ));
}

void alignCenter() {

}

void release() {
    if (d_psfDev != nullptr) {
        cudaErrChk(hipFreeArray(d_psfDev));
        // unbind the texture
        cudaErrChk(hipUnbindTexture(psfTexRef));
    }
}

}

namespace Kernel {

texture<hipfftComplex, 2, hipReadModeElementType> otfTex;

template <typename T_out, typename T_in>
__global__
void convertTypeKernel(T_out *dst, T_in *src,
                       const int nx, const int ny, const int nz,
                       const size_t pitchDst, const size_t pitchSrc) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < nx * ny * nz;
         i += blockDim.x * gridDim.x) {
        int z = i / nx / ny;
        int y = (i / nx) % ny;
        int x = i % nx;
        int nPitchSrc = pitchSrc / sizeof(T_in);
        int nPitchDst = pitchDst / sizeof(T_out);
        T_in* ptrSrc = src + (z * ny + y) * nPitchSrc + x;
        T_out* ptrDst = dst + (z * ny + y) * nPitchDst + x;
        *ptrDst = (T_out)(*ptrSrc);
    }
}

template <typename T_out, typename T_in>
__host__
void convertType(T_out *dst, T_in *src,
                 const hipExtent extDst, const hipExtent extSrc) {
    int nSMs;
    hipDeviceGetAttribute(&nSMs, hipDeviceAttributeMultiprocessorCount, 0);
    int nx = extSrc.width / sizeof(T_in);
    int ny = extSrc.height;
    int nz = extSrc.depth;
    convertTypeKernel<T_out, T_in><<<32*nSMs, 256>>>(dst, src, nx, ny, nz,
                                                     extDst.width, extSrc.width);
}

/*
 * Explicit instantiation
 */
template void convertType(hipfftReal *dst, uint16_t *src,
                          const hipExtent extDst, const hipExtent extSrc);

}
