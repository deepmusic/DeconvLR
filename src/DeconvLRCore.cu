#include "hip/hip_runtime.h"
// corresponded header file
// necessary project headers
#include "DeconvLRCore.cuh"
// 3rd party libraries headers
#include <hipfft/hipfft.h>
// standard libraries headers
// system headers

namespace Kernel {

inline int iDivUp(int a, int b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

template <typename T_out, typename T_in>
__global__
void convertTypeKernel(T_out *dst, T_in *src, const hipExtent size) {

}

template <typename T_out, typename T_in>
__host__
void convertType(T_out *dst, T_in *src, const hipExtent size) {
    int nSMs;
    hipDeviceGetAttribute(&nSMs, hipDeviceAttributeMultiprocessorCount, 0);

    convertTypeKernel<T_out, T_in><<<32*nSMs, 256>>>(dst, src, size);
}

// explicit instantiation
template void convertType(hipfftReal *dst, uint16_t *src, const hipExtent size);

}
