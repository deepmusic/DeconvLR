#include "hip/hip_runtime.h"
// corresponded header file
// necessary project headers
#include "DeconvLRCore.cuh"
#include "Helper.cuh"
// 3rd party libraries headers
#include <hipfft/hipfft.h>
// standard libraries headers
#include <cstdint>
// system headers

namespace PSF {

hipArray_t d_psf = nullptr;
texture<texture, 3, hipReadModeElementType> psfTex;

void bindData(
    const float *h_psf,
    const size_t nx, const size_t ny, const size_t nz
) {
    // create hipArray for the texture
    hipChannelFormatDesc desc = hipCreateChannelDesc(
        32, 0, 0, 0, hipChannelFormatKindFloat
    );
    hipExtent extent = make_hipExtent(
        nx, ny, nz
    );
    cudaErrChk(hipMalloc3DArray(
        &d_psf,
        &desc,
        extent,
        hipArrayDefault
    ));

    // copy data from host to device
    hipMemcpy3DParms parms = {0};
    parms.srcPtr = make_hipPitchedPtr(
        h_psf,
        nx * sizeof(float), nx, ny
    ),
    parms.dstArray = d_psf,
    parms.extent = extent,
    parms.kind = hipMemcpyHostToDevice
    cudaErrChk(hipMemcpy3D(&parms));

    // bind the texture
    cudaErrChk(hipBindTextureToArray(
        psfTex,     // texture to bind
        d_psf,      // memory array on device
        &desc       // channel format
    ));
}

void findCenter(float *cx, float *cy, float *cz) {

}

void alignCenter() {

}

void release() {
    if (d_psf != nullptr) {
        cudaErrChk(hipFreeArray(d_psf));
        // unbind the texture
        cudaErrChk(hipUnbindTexture(psfTex));
    }
}

}

namespace Kernel {

texture<hipfftComplex, 2, hipReadModeElementType> otfTex;

__host__
void interpolateOTF() {

}

inline int iDivUp(int a, int b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

template <typename T_out, typename T_in>
__global__
void convertTypeKernel(T_out *dst, T_in *src,
                       const int nx, const int ny, const int nz,
                       const size_t pitchDst, const size_t pitchSrc) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < nx * ny * nz;
         i += blockDim.x * gridDim.x) {
        int z = i / nx / ny;
        int y = (i / nx) % ny;
        int x = i % nx;
        int nPitchSrc = pitchSrc / sizeof(T_in);
        int nPitchDst = pitchDst / sizeof(T_out);
        T_in* ptrSrc = src + (z * ny + y) * nPitchSrc + x;
        T_out* ptrDst = dst + (z * ny + y) * nPitchDst + x;
        *ptrDst = (T_out)(*ptrSrc);
    }
}

template <typename T_out, typename T_in>
__host__
void convertType(T_out *dst, T_in *src,
                 const hipExtent extDst, const hipExtent extSrc) {
    int nSMs;
    hipDeviceGetAttribute(&nSMs, hipDeviceAttributeMultiprocessorCount, 0);
    int nx = extSrc.width / sizeof(T_in);
    int ny = extSrc.height;
    int nz = extSrc.depth;
    convertTypeKernel<T_out, T_in><<<32*nSMs, 256>>>(dst, src, nx, ny, nz,
                                                     extDst.width, extSrc.width);
}

/*
 * Explicit instantiation
 */
template void convertType(hipfftReal *dst, uint16_t *src,
                          const hipExtent extDst, const hipExtent extSrc);

}
