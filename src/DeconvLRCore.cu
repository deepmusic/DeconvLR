#include "hip/hip_runtime.h"
// corresponded header file
// necessary project headers
#include "DeconvLRCore.cuh"
// 3rd party libraries headers
#include <hip/hip_runtime.h>
// standard libraries headers
#include <iostream>
// system headers

// Kernel that executes on the CUDA device
__global__
void square_array_kernel(float *a, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) {
      a[idx] = a[idx] * a[idx];
  }
}

__host__
void square_array(float *a, int N) {
    int block_size = 4;
    int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
    std::cout << "running kernel" << std::endl;
    square_array_kernel<<<n_blocks, block_size>>>(a, N);
}
