#include "hip/hip_runtime.h"
// corresponded header file
// necessary project headers
#include "DeconvLRCore.cuh"
// 3rd party libraries headers
#include <hipfft/hipfft.h>
// standard libraries headers
#include <cstdint>
// system headers

namespace Kernel {

texture<hipfftComplex, 2, hipReadModeElementType> otfTex;

__host__
void interpolateOTF() {

}

inline int iDivUp(int a, int b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

template <typename T_out, typename T_in>
__global__
void convertTypeKernel(T_out *dst, T_in *src, const hipExtent size) {
    //TODO fill the blank here, type convert form T_in to T_out
}

template <typename T_out, typename T_in>
__host__
void convertType(T_out *dst, T_in *src, const hipExtent size) {
    int nSMs;
    hipDeviceGetAttribute(&nSMs, hipDeviceAttributeMultiprocessorCount, 0);

    convertTypeKernel<T_out, T_in><<<32*nSMs, 256>>>(dst, src, size);
}

/*
 * Explicit instantiation
 */
template void convertType(hipfftReal *dst, uint16_t *src, const hipExtent size);

}
