// corresponded header file
// necessary project headers
#include "DeconvRLImpl.cuh"
#include "Helper.cuh"
// 3rd party libraries headers
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hipfft/hipfft.h>
// standard libraries headers
#include <cstdint>
// system headers

namespace DeconvRL {

namespace Core {

namespace RL {

enum class ConvType {
    PLAIN = 1, CONJUGATE
};

namespace {
// generic complex number operation
template <ConvType type>
struct MultiplyAndScale
    : public thrust::binary_function<hipComplex, hipComplex, hipComplex> {
    MultiplyAndScale(const float c_)
        : c(c_) {
        if (type == ConvType::CONJUGATE) {
            printf("CONJUGATE\n");
        } else {
            printf("PLAIN\n");
        }
    }

    __host__ __device__
    hipComplex operator()(const hipComplex &a, const hipComplex &b) const {
        if (type == ConvType::CONJUGATE) {
            return hipCmulf(a, hipConjf(b))/c;
        } else {
            return hipCmulf(a, b)/c;
        }
    }

private:
    const float c;
};

template <ConvType type>
void filter(
    hipfftReal *odata, const hipfftReal *idata, const hipfftComplex *otf,
    Core::RL::Parameters &parm
) {
    fprintf(stderr, "[DEBUG] +++ ENTER RL::(anon)::filter() +++\n");

    const size_t nelem = (parm.nx/2+1) * parm.ny * parm.nz;
    hipfftComplex *buffer = (hipfftComplex *)parm.filterBuffer.complexA;

    // convert to frequency space
    cudaErrChk(hipfftExecR2C(
        parm.fftHandle.forward,
        const_cast<hipfftReal *>(idata),
        buffer
    ));

    // element-wise multiplication and scale down
    thrust::transform(
        thrust::device,
        buffer, buffer+nelem,       // first input sequence
        otf,                        // second input sequence
        buffer,                     // output sequence
        MultiplyAndScale<type>(1.0f/nelem)
    );

    // convert back to real space
    cudaErrChk(hipfftExecC2R(
        parm.fftHandle.reverse,
        buffer,
        odata
    ));

    fprintf(stderr, "[DEBUG] +++ EXIT RL::(anon)::filter() +++\n");
}

thrust::divides<float> DivfOp;
thrust::multiplies<float> MulfOp;

}

void step(
    float *odata, const float *idata,
    Core::RL::Parameters &parms
) {
    fprintf(stderr, "[DEBUG] +++ ENTER RL::step() +++\n");

    const size_t nelem = parms.nelem;
    hipfftReal *buffer = parms.RLBuffer.realA;

    hipfftComplex *otf = parms.otf;

    /*
     * \hat{f_{k+1}} =
     *     \hat{f_k} \left(
     *         h \ast \frac{g}{h \otimes \hat{f_k}}
     *     \right)
     */

    // reblur the image
    fprintf(stderr, "A\n");
    filter<ConvType::PLAIN>(buffer, idata, otf, parms);
    fprintf(stderr, "B\n");
    // error
    thrust::transform(
        thrust::device,
        parms.raw,  parms.raw+nelem,
        buffer,
        buffer, // output
        DivfOp
    );
    fprintf(stderr, "C\n");
    filter<ConvType::CONJUGATE>(buffer, buffer, otf, parms);
    fprintf(stderr, "D\n");
    // latent image
    thrust::transform(
        thrust::device,
        idata, idata+nelem,
        buffer,
        odata,  // output
        MulfOp
    );
    fprintf(stderr, "E\n");

    fprintf(stderr, "[DEBUG] +++ EXIT RL::step() +++\n");
}

}

namespace Biggs {

namespace {

}

void step(
    float *odata, const float *idata,
    Core::RL::Parameters &parm
) {
    // execute an iteration of RL
    //RL::step();

    // find the update direction

    // calculate acceleration factor

    // re-estimate prediction
}

}

}

namespace Common {

namespace {

template <typename T>
struct ToFloat
    : public thrust::unary_function<const T, float> {
    __host__ __device__
    float operator()(const T &v) const {
        return (float)v;
    }
};

}

void ushort2float(float *odata, const uint16_t *idata, const size_t nelem) {
    thrust::transform(
        thrust::device,
        idata, idata + nelem,   // input
        odata,                  // output
        ToFloat<uint16_t>()
    );
}

}

}
